#include "hip/hip_runtime.h"
// PCOL_OMP_C99.cpp : Defines the entry point for the console application.
//
// A GPU accelerated in-house code which solves the steady Navier-Stokes equations,
// using the Finite Volume Method (FVM) in primitive variables formulation.
// Convective terms are discretized using second order central differencing scheme,
// and SIMPLE algorithm are used to decouple velocity and pressure. 
// Strongly Implicit Procedure was used to solve the resulted linear algebraic equations.

// Copyright 2016 Tamer A. Abdelmigid

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <algorithm>
#include <iomanip>
#include <Windows.h>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;



__global__ void Forwardsub(double *RES, double *LS, double *LW, double *LPR, int NI, int NJ, int Start, int J, int threads)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < threads)
	{
		int IJ = ((Start + i)*NI) + (J - (Start + i));
		//printf(" %d ", IJ);
		RES[IJ] = (RES[IJ] - LS[IJ] * RES[IJ - 1] - LW[IJ] * RES[IJ - NJ])*LPR[IJ];
	}

}

__global__ void Forward(double *D_RES, double *D_LS, double *D_LW, double *D_LPR, int NI, int NJ, int thrds)
{
	int threads;
	for (int J = 2; J <= (NI + NI - 2); ++J)
	{
		int Start = max(1, J - NI + 2);
		int End = min(NI - 2, J - 1);

		threads = End - Start + 1;
		dim3 block(thrds);
		dim3 grid((threads + block.x - 1) / block.x);

		if ((End - Start + 1) > 0)
		{
			Forwardsub << <grid, block >> >(D_RES, D_LS, D_LW, D_LPR, NI, NJ, Start, J, threads);
			hipDeviceSynchronize();
		}
	}
}

__global__ void Backwardsub(double *U, double *RES, double *UN, double *UE, double *LPR, int NI, int NJ, int End, int J, int threads)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < threads)
	{
		int IJ = ((End - i)*NI) + (J - (End - i));
		RES[IJ] = RES[IJ] - UN[IJ] * RES[IJ + 1] - UE[IJ] * RES[IJ + NJ];
		U[IJ] = U[IJ] + RES[IJ];
	}
}

__global__ void Backward(double *D_U, double *D_RES, double *D_UN, double *D_UE, double *D_LPR, int NI, int NJ, int thrds)
{
	int threads;
	for (int J = (NI + NI - 2); J >= 2; --J)
	{
		int Start = max(1, J - NI + 2);
		int End = min(NI - 2, J - 1);

		threads = End - Start + 1;
		dim3 block(thrds);
		dim3 grid((threads + block.x - 1) / block.x);

		if ((End - Start + 1) > 0)
		{
			Backwardsub << <grid, block >> >(D_U, D_RES, D_UN, D_UE, D_LPR, NI, NJ, End, J, threads);
			hipDeviceSynchronize();
		}
	}
}

__global__ void UpdateU(double *U, double *RES)
{
	int IJ = threadIdx.x;
	U[IJ] = U[IJ] + RES[IJ];

}

int main()
{

	////////////////
	//  Read Data //
	////////////////

	string Ofilename, DAT;
	ifstream INPUTFILE;
	INPUTFILE.open("Input.txt");


	int thrds = 64;
	int bkthrds = 32;


	double XMIN, XMAX;					// Positional limits of the solution domain
	double EXX;							// EXX: Grid Expansion Factor in x-direction
	int N = 101;								// NI: No. of control volumes in x-direction
	int NI;							// NIM: No. of nodes is x-direction
	int NIM;
	double DX;							// DX: Distance between two consecutive nodes
	double YMIN, YMAX;					// Positional limits of the solution domain
	double EXY;							// EXY: Grid Expansion Factor in y-direction
	int M = 101;								// NJ: No. of control volumes in y-direction
	int NJ;							// NJM: No. of nodes is y-direction
	int NJM;
	double DY;							// DY: Distance between two consecutive nodes

	double DEN = 1;							// DEN: Density
	double VIS = 0.001;							// VIS: Dynamic Viscosity
	double ULID = 1;						// ULID: Lid Velocity
	int MAXIT = 100;                          // MAXIT: Maximum No. of Outer Iteration
	int MAXITU = 1;                         // MAXITU: Maximum No. of Inner U velocity Iterations
	int MAXITV = 1;                         // MAXITU: Maximum No. of Inner V velocity Iterations
	int MAXITP = 6;                         // MAXITU: Maximum No. of Inner P Pressure Iterations
	int MAXITT = 1;                         // MAXITU: Maximum No. of Inner T Pressure Iterations
	double RESMAX = 0.00001;						// RESMAX: Maximum residual at which outer iterations to be stopped
	double RESMAXU = 0.2;						// RESMAXU: Maximum residual at which inner U iterations to be stopped
	double RESMAXV = 0.2;						// RESMAXV: Maximum residual at which inner V iterations to be stopped
	double RESMAXP = 0.01;						// RESMAXP: Maximum residual at which inner P iterations to be stopped
	double RESMAXT = 0.2;						// RESMAXP: Maximum residual at which inner T iterations to be stopped
	double SLARGE;						// SLARGE: Divergence Criteria
	double URFU = 0.8;						// URFU: Under-relaxation factor for U-velocity
	double URFV = 0.8;						// URFV: Under-relaxation factor for V-velocity
	double URFP = 0.3;						// URFP: Under-relaxation factor for P-Pressure
	double URFT = 0.9;						// URFP: Under-relaxation factor for T-Temperature
	double ALFA = 0.93;						// ALFA: Value needed by SIP Solver
	int IMON, JMON;						// IMON, JMON: X and Y coordinates of a point to monitor its values
	double PRM;							// PRM: Fluid Prandtl Number
	double GRAVX, GRAVY;				// GRAVX, GRAVY: Gravity in X and Y direction respectively
	double BETA;						// BETA: Fluid Volumetric expansion factor
	double TH, TC, TREF;				// TH, TC, TREF: Hot temperature, Cold temperature, and Reference Temperature
	int IPR, JPR;						// IPR, JPR: I and J indices of Pressure reference point
	bool LTIME;							// LTIME: True means Unsteady Simulation
	bool LCALIEN;						// LCALIEN: True Means Energy equation is solved
	double ITST;						// ITST: the number of time steps to be performed 
	// (1 if steady flow is considered)
	double NPRT;						// NPRT: Number of steps after which solution is saved
	double DT;							// DT: Time step size
	double TPER;						// TPER: The oscillation period in the case of unsteady flow
	// with oscillating lid.
	double GAMT;						// GAMT: the blending factor for time differencing schemes
	// (GAMT = 1->three time levels scheme, GAMT = 0->Euler implicit scheme).
	double GDSU, GDSV, GDSP, GDST;		// GDSU, GDSV, GDSP, GDST: the blending factor for UDS and CDS 
	// in the equation for U, V, P, T respectively 
	// (convective terms; value 1.0 means CDS(second order)
	// 0.0 means UDS(first order), any value between 0.0 and 1.0 can be used).
	// The value 1.0 is recommended, except for coarse grids,
	// in case convergence problems are encountered.
	double TIME;						// TIME: Time step
	double SOURCE, RESOR, RSMU, RSMV, RSMP, RSMT;
	RSMU = RSMV = RSMP = RSMT = 0.0;
	double progress = 0.0;
	string SLTIME, SLCALIEN;
	string InterSave, InterRead;
	int InterSaveNo;

	XMIN = YMIN = 0;
	XMAX = YMAX = 1;


	/* Output File Name */
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE >> Ofilename;

	ofstream outputfile;
	ofstream resultfile;
	fstream Intermediate;
	outputfile.open(string(Ofilename + ".txt").c_str());


	/* Generate "ASCII" .dat File ? */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> DAT;

	if (DAT == "True" || DAT == "true")
	{
		resultfile.open(string(Ofilename + ".dat").c_str());
	}

	/* GRID */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> N >> M;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> EXX >> EXY;


	/* Discritization Settings */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> GDSU >> GDSV >> GDSP >> GDST;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> GAMT;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> URFU >> URFV >> URFP >> URFT;


	/* Fluid Properties */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> DEN;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> VIS;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> BETA;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> PRM;


	/* Control */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> SLCALIEN;

	if (SLCALIEN == "True" || SLCALIEN == "true")
	{
		LCALIEN = true;
	}

	else
	{
		LCALIEN = false;
	}

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> SLTIME;

	if (SLTIME == "True" || SLTIME == "true")
	{
		LTIME = true;
	}

	else
	{
		LTIME = false;
	}


	/* Simulation Paramters */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> ULID;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> GRAVX;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> GRAVY;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> TH;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> TC;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> TREF;


	/* Time Controls */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> ITST;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> NPRT;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> DT;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> TPER;


	/* Monitoring Options */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> IMON >> JMON;


	/* Solver Criteria */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> MAXIT;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> MAXITU >> MAXITV >> MAXITP >> MAXITT;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> RESMAX;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> RESMAXU >> RESMAXV >> RESMAXP >> RESMAXT;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> SLARGE;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> ALFA;

	/* Intermediate Save */
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, ':');
	INPUTFILE.ignore(1024, '/');
	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> InterSave;

	if (InterSave == "True" || InterSave == "true")
	{

	}

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> InterSaveNo;

	INPUTFILE.ignore(1024, '|');
	INPUTFILE >> InterRead;

	/////////////////////
	// Define The Grid //
	/////////////////////



	/**************************************************************/

	NI = N + 2;
	NIM = NI - 1;

	double *X;
	X = (double*)malloc(NI*sizeof(double));			// X: Vector storing the X-position of nodes

	if (EXX == 1)
		DX = (XMAX - XMIN) / N;
	else
		DX = (XMAX - XMIN)*(1.0 - EXX) / (1 - pow(EXX, N));


	X[0] = XMIN;
	for (int i = 1; i < NIM; ++i)
	{
		X[i] = X[i - 1] + DX;
		DX = DX*EXX;
	}
	X[NI - 1] = X[NIM - 1];

	/* Coordinates of CV centers in x-direction */

	double *XC;
	XC = (double*)malloc(NI*sizeof(double));			// XC: Vector storing the X-position of cell centers


	XC[0] = X[0];
	for (int i = 1; i < NIM; ++i)
	{
		XC[i] = 0.5*(X[i] + X[i - 1]);
	}
	XC[NI - 1] = X[NIM - 1];

	/*for (int i = 0; i < NI; ++i)
	{
	outputfile << XC(i) << endl;
	}*/

	/**************************************************************/

	/* Define Y-Grid */


	NJ = M + 2;
	NJM = NJ - 1;

	double *Y;
	Y = (double*)malloc(NJ*sizeof(double));			// Y: Vector storing the Y-position of nodes



	if (EXY == 1)
		DY = (YMAX - YMIN) / M;
	else
		DY = (YMAX - YMIN)*(1.0 - EXY) / (1 - pow(EXY, M));


	Y[0] = YMIN;
	for (int i = 1; i < NJM; ++i)
	{
		Y[i] = Y[i - 1] + DY;
		DY = DY*EXY;
	}
	Y[NJ - 1] = Y[NJM - 1];



	/* Coordinates of CV centers in y-direction */

	double *YC;
	YC = (double*)malloc(NJ*sizeof(double));			// YC: Vector storing the Y-position of cell centers		

	YC[0] = Y[0];
	for (int j = 1; j < NJM; ++j)
	{
		YC[j] = 0.5*(Y[j] + Y[j - 1]);
	}
	YC[NJ - 1] = Y[NJM - 1];


	/**************************************************************/

	/* Set Radius R=1 for Plane */

	double *R;
	R = (double*)malloc(NJ*sizeof(double));

	for (int j = 0; j < NJ; ++j)
	{
		R[j] = 1.0;
	}

	/**************************************************************/

	/* INTERPOLATION FACTORS (ON SCALAR CVs) */

	double *FX, *FY;
	FX = (double*)malloc(NI*sizeof(double));
	FY = (double*)malloc(NJ*sizeof(double));


	FX[0] = 0.0;
	FY[0] = 0.0;
	FX[NI - 1] = 0.0;
	FY[NJ - 1] = 0.0;

	for (int I = 1; I < NIM; ++I)
	{
		FX[I] = (X[I] - X[I - 1]) / (X[I + 1] - X[I - 1]);
	}

	for (int J = 1; J < NIM; ++J)
	{
		FY[J] = (Y[J] - Y[J - 1]) / (Y[J + 1] - Y[J - 1]);
	}


	//////////////////////////////////////////////////////////////////////////////////////////////////////////

	/////////////////////////
	// Data Initialization //
	/////////////////////////




	LARGE_INTEGER StartingTimeALL, EndingTimeALL, ElapsedMicrosecondsALL;
	LARGE_INTEGER StartingTimeU, EndingTimeU, ElapsedMicrosecondsU;
	LARGE_INTEGER StartingTimeV, EndingTimeV, ElapsedMicrosecondsV;
	LARGE_INTEGER StartingTimeP, EndingTimeP, ElapsedMicrosecondsP;
	LARGE_INTEGER StartingTimeT, EndingTimeT, ElapsedMicrosecondsT;

	LARGE_INTEGER StartingTimeTU, EndingTimeTU, ElapsedMicrosecondsTU;
	LARGE_INTEGER StartingTimeTV, EndingTimeTV, ElapsedMicrosecondsTV;
	LARGE_INTEGER StartingTimeTP, EndingTimeTP, ElapsedMicrosecondsTP;
	LARGE_INTEGER StartingTimeTT, EndingTimeTT, ElapsedMicrosecondsTT;

	LARGE_INTEGER StartingTimeSU, EndingTimeSU, ElapsedMicrosecondsSU;
	LARGE_INTEGER StartingTimeSV, EndingTimeSV, ElapsedMicrosecondsSV;
	LARGE_INTEGER StartingTimeSP, EndingTimeSP, ElapsedMicrosecondsSP;
	LARGE_INTEGER StartingTimeST, EndingTimeST, ElapsedMicrosecondsST;

	LARGE_INTEGER Frequency;
	double TIMEU, TIMEV, TIMEP, TIMET;
	TIMEU = TIMEV = TIMEP = TIMET = 0.0;

	double TIMETU, TIMETV, TIMETP, TIMETT;
	TIMETU = TIMETV = TIMETP = TIMETT = 0.0;

	double TIMESU, TIMESV, TIMESP, TIMEST;
	TIMESU = TIMESV = TIMESP = TIMEST = 0.0;


	double SMALL = 0.000000000000000000000000000001;
	double LARGE = 100000000000000000;

	int IJMON;




	/* For Steady Lid Driven Cavity */

	//GRAVX = GRAVY = TH = TC = TREF = BETA = 0.0;
	//PRM = TPER = 1.0;
	//ITST = NPRT = 1.0;
	//DT = LARGE;
	//GAMT = 1.0;
	//GDSU = GDSV = GDSP = GDST = 1.0;
	//LTIME = false;
	//LCALIEN = false;
	IPR = JPR = 1;

	double PRR = 1.0 / PRM;
	double OM = 8.0 * atan(1.0) / TPER;
	double DTR = 1.0 / DT;
	double URFUR, URFVR, URFPR, URFTR;
	URFPR = 1.0 / URFP;

	TIME = 0.0;

	///////////////////////////////////////////////////////////////////////////////////////////////

	int *LI;
	LI = (int*)malloc(NJ*sizeof(int));

	for (int I = 0; I < NI; ++I)
	{
		LI[I] = I*NJ;
	}

	IJMON = LI[IMON] + JMON;


	double NIJ = NI*NJ;

	double *U, *V, *P, *T;
	U = (double*)malloc(NIJ*sizeof(double));
	V = (double*)malloc(NIJ*sizeof(double));
	P = (double*)malloc(NIJ*sizeof(double));
	T = (double*)malloc(NIJ*sizeof(double));


	for (int i = 0; i < NIJ; ++i)
	{
		U[i] = V[i] = P[i] = T[i] = 0.0;
	}

	double *UO, *VO, *PO, *TO;
	UO = (double*)malloc(NIJ*sizeof(double));
	VO = (double*)malloc(NIJ*sizeof(double));
	PO = (double*)malloc(NIJ*sizeof(double));
	TO = (double*)malloc(NIJ*sizeof(double));

	double *UOO, *VOO, *POO, *TOO;
	UOO = (double*)malloc(NIJ*sizeof(double));
	VOO = (double*)malloc(NIJ*sizeof(double));
	POO = (double*)malloc(NIJ*sizeof(double));
	TOO = (double*)malloc(NIJ*sizeof(double));

	double *AN, *AS, *AE, *AW, *AP;
	AN = (double*)malloc(NIJ*sizeof(double));
	AS = (double*)malloc(NIJ*sizeof(double));
	AE = (double*)malloc(NIJ*sizeof(double));
	AW = (double*)malloc(NIJ*sizeof(double));
	AP = (double*)malloc(NIJ*sizeof(double));

	double *UE, *UN, *LW, *LS, *LES, *RES, *LPR;
	UE = (double*)malloc(NIJ*sizeof(double));
	UN = (double*)malloc(NIJ*sizeof(double));
	LW = (double*)malloc(NIJ*sizeof(double));
	LS = (double*)malloc(NIJ*sizeof(double));
	LES = (double*)malloc(NIJ*sizeof(double));
	RES = (double*)malloc(NIJ*sizeof(double));
	LPR = (double*)malloc(NIJ*sizeof(double));

	double P1, P2, RESL, RSM;


	double  *D_RES, *D_LS, *D_LW, *D_UN, *D_UE, *D_LPR;

	hipMalloc(&D_RES, NIJ*sizeof(double));
	hipMalloc(&D_LS, NIJ*sizeof(double));
	hipMalloc(&D_UN, NIJ*sizeof(double));
	hipMalloc(&D_UE, NIJ*sizeof(double));
	hipMalloc(&D_LW, NIJ*sizeof(double));
	hipMalloc(&D_LPR, NIJ*sizeof(double));

	double  *D_U, *D_V, *D_PP, *D_T;

	hipMalloc(&D_U, NIJ*sizeof(double));
	hipMalloc(&D_V, NIJ*sizeof(double));
	hipMalloc(&D_PP, NIJ*sizeof(double));
	hipMalloc(&D_T, NIJ*sizeof(double));


	/*******************************************/

	double FXE, FXP, DXPE, S, D, CE, CP, CN;
	double FYN, FYP, DYPN;
	double FUUDS, FVUDS, FUCDS, FVCDS;
	double RP, VOL;
	double PE, PW, PS, PN, SB;
	double APT;

	double *F1, *F2, *DPX, *DPY;
	F1 = (double*)malloc(NIJ*sizeof(double));
	F2 = (double*)malloc(NIJ*sizeof(double));
	DPX = (double*)malloc(NIJ*sizeof(double));
	DPY = (double*)malloc(NIJ*sizeof(double));

	/*******************************************/

	double *SU, *SV, *APU, *APV;
	SU = (double*)malloc(NIJ*sizeof(double));
	SV = (double*)malloc(NIJ*sizeof(double));
	APU = (double*)malloc(NIJ*sizeof(double));
	APV = (double*)malloc(NIJ*sizeof(double));

	/*******************************************/

	double VOLE, DPXEL, UEL, APUE, DPXE, UEP;
	double VOLN, DPYNL, VNL, APVN, DPYN, VN;
	double PPO;
	int IJPREF;
	double PPE, PPW, PPN, PPS;

	double SUM;

	double *PP;
	PP = (double*)malloc(NIJ*sizeof(double));

	/*******************************************/

	double FUDS, FCDS;


	/////////////////////////////////////
	// BOUNDARY AND INITIAL CONDITIONS //
	/////////////////////////////////////

	/* WEST AND EAST ISOTHERMAL BOUNDARIES */

	for (int J = 0; J < NJ; ++J)
	{
		T[J] = TH;
	}

	for (int J = 0; J < NJ; ++J)
	{
		T[(LI[NI - 1] + J)] = TC;
	}

	/* NORTH WALL VELOCITY (FOR LID-DRIVEN CAVITY) */

	if (LTIME)
	{
		for (int I = 1; I < NIM; ++I)
		{
			int IJ = LI[I] + NJ - 1;
			U[IJ] = ULID*sin(OM*TIME);
		}
	}

	else
	{
		for (int I = 1; I < NIM; ++I)
		{
			int IJ = LI[I] + NJ - 1;
			U[IJ] = ULID;
		}
	}

	//////////////////////////////////////////////////////////////////////////////////////////////////////

	outputfile << "                               PREDICTION OF FLOW IN LID DRIVEN CAVITIES " << endl;
	outputfile << "                               ========================================= " << endl;
	outputfile << "                               Reynolds Number     : "
		<< setw(5) << scientific << setprecision(3) << left << setfill('0') << (ULID*DEN*1.0) / VIS << endl;
	outputfile << "                               Prandtl Number      : "
		<< setw(5) << scientific << setprecision(3) << left << setfill('0') << PRM << endl;
	outputfile << "                               Fluid Density       : "
		<< setw(5) << scientific << setprecision(3) << left << setfill('0') << DEN << endl;
	outputfile << "                               Dynamic Viscosity   : "
		<< setw(5) << scientific << setprecision(3) << left << setfill('0') << VIS << endl;
	outputfile << endl << endl;

	outputfile << "                               Gravity in X-Dir.     : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GRAVX << endl;
	outputfile << "                               Gravity in Y-Dir.     : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GRAVY << endl;
	outputfile << "                               Hot Wall Temperature  : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << TH << endl;
	outputfile << "                               Cold Wall Temperature : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << TC << endl;
	outputfile << endl << endl;

	outputfile << "                               U - Under Relaxation Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << URFU << endl;
	outputfile << "                               V - Under Relaxation Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << URFV << endl;
	outputfile << "                               P - Under Relaxation Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << URFP << endl;
	outputfile << "                               T - Under Relaxation Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << URFT << endl;
	outputfile << endl << endl;

	outputfile << "                               U - Blending Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GDSU << endl;
	outputfile << "                               V - Blending Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GDSV << endl;
	outputfile << "                               P - Blending Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GDSP << endl;
	outputfile << "                               T - Blending Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GDST << endl;
	outputfile << endl << endl;

	outputfile << "                               NO of Nodes in X-Direction : "
		<< setw(3) << fixed << left << setfill(' ') << N << endl;
	outputfile << "                               NO of Nodes in Y-Direction : "
		<< setw(3) << fixed << left << setfill(' ') << M << endl;
	outputfile << endl << endl;

	outputfile << "                               Alfa Parameter : "
		<< setw(3) << fixed << left << setfill(' ') << ALFA << endl;
	outputfile << endl << endl;

	//////////////////////////////////////////////////////////////////////////////////////////////////////////////

	cout << "                    PREDICTION OF FLOW IN LID DRIVEN CAVITIES " << endl;
	cout << "                    ========================================= " << endl;
	cout << "                    Reynolds Number     : "
		<< setw(5) << scientific << setprecision(3) << left << setfill('0') << (ULID*DEN*1.0) / VIS << endl;
	cout << "                    Prandtl Number      : "
		<< setw(5) << scientific << setprecision(3) << left << setfill('0') << PRM << endl;
	cout << "                    Fluid Density       : "
		<< setw(5) << scientific << setprecision(3) << left << setfill('0') << DEN << endl;
	cout << "                    Dynamic Viscosity   : "
		<< setw(5) << scientific << setprecision(3) << left << setfill('0') << VIS << endl;
	cout << endl;

	cout << "                    U - Under Relaxation Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << URFU << endl;
	cout << "                    V - Under Relaxation Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << URFV << endl;
	cout << "                    P - Under Relaxation Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << URFP << endl;
	cout << "                    T - Under Relaxation Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << URFT << endl;
	cout << endl;

	cout << "                    U - Blending Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GDSU << endl;
	cout << "                    V - Blending Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GDSV << endl;
	cout << "                    P - Blending Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GDSP << endl;
	cout << "                    T - Blending Factor : "
		<< setw(5) << fixed << setprecision(2) << left << setfill(' ') << GDST << endl;
	cout << endl;

	cout << "                    NO of Nodes in X-Direction : "
		<< setw(3) << fixed << left << setfill(' ') << N << endl;
	cout << "                    NO of Nodes in Y-Direction : "
		<< setw(3) << fixed << left << setfill(' ') << M << endl;
	cout << endl;

	cout << "                    Alfa Parameter : "
		<< setw(3) << fixed << left << setfill(' ') << ALFA << endl;
	cout << endl;


	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


	/****************************************************/

	QueryPerformanceFrequency(&Frequency);
	QueryPerformanceCounter(&StartingTimeALL);

	/***************************************************/


	///////////////
	// Time Loop //
	///////////////

	double ITIMS = 0.0;
	double ITIME = ITIMS + ITST;

	for (double ITIM = ITIMS; ITIM < ITIME; ++ITIM)
	{
		TIME = TIME + DT;

		/* SHIFT SOLUTIONS IN TIME */

		if (LTIME)
		{
			for (int IJ = 0; IJ < NIJ; ++IJ)
			{
				TOO[IJ] = TO[IJ];
				UOO[IJ] = UO[IJ];
				VOO[IJ] = VO[IJ];
				TO[IJ] = T[IJ];
				UO[IJ] = U[IJ];
				VO[IJ] = V[IJ];
			}
		}

		outputfile << "TIME = " << TIME << endl;
		outputfile << "===================" << endl;
		outputfile << "ITER    I-------------ABSOLUTE RESIDUAL SOURCE SUMS-------------I      I---------FIELD VALUES AT LOCATION ("
			<< fixed << left << IMON << "," << fixed << left << JMON << ")---------I" << endl;
		outputfile << "--------------------------------------------------------------------------------------------------------------------------------------------" << endl;
		outputfile << "NO     |     RSMU     |     RSMV     |     RSMP     |     RSMT     ||       UMON       |       VMON       |      PMON       |     TMON" << endl;
		outputfile << "--------------------------------------------------------------------------------------------------------------------------------------------" << endl;


		if (LTIME)
		{
			for (int I = 1; I < NIM; ++I)
			{
				int IJ = LI[I] + NJ - 1;
				U[IJ] = ULID*sin(OM*TIME);
			}
		}

		///////////////////////////////////////////
		// OUTER ITERATIONS (SIMPLE RELAXATIONS) //
		///////////////////////////////////////////

		for (int ITER = 0; ITER < MAXIT; ++ITER)
		{

			if (InterRead == "True" || InterRead == "true")
			{
				if (ITER == 0)
				{
					Intermediate.open(string(Ofilename + ".inter").c_str(), ios::in);
					Intermediate >> ITER;
					ITER = 0;
					while (!Intermediate.eof())
					{
						int I, J;
						Intermediate >> I >> J;
						int IJ = LI[I] + J;
						Intermediate >> U[IJ] >> V[IJ] >> P[IJ] >> T[IJ];
					}
					Intermediate.close();
				}
			}

			////////////
			// CALCUV //
			////////////




			/* This routine sets the coefficient matrix for the U and
			V equations, and calls the linear equation solver to
			update the velocity components. Constant fluid
			properties are assumed */

			/* RECIPROCAL VALUES OF UNDER-RELAXATION FACTORS FOR U AND V */

			URFUR = 1.0 / URFU;
			URFVR = 1.0 / URFV;

			/* SET BOUNDARY PRESSURE (LINEAR EXTRAPOLATION FROM INSIDE) */

			/*--------------------------------------------------------------------------------*/

			/* PBOUND */

			/* This routine calculates boundary values of pressure or
			pressure-correction by extrapolating (linearly) from inside.*/

			/* SOUTH AND NORTH BOUNDARIES */

			for (int I = 1; I < NIM; ++I)
			{
				int IJ = LI[I];
				P[IJ] = P[IJ + 1] + (P[IJ + 1] - P[IJ + 2])*FY[1];
				IJ = LI[I] + NJ - 1;
				P[IJ] = P[IJ - 1] + (P[IJ - 1] - P[IJ - 2])*(1.0 - FY[NJM - 1 - 1]);
			}

			/* WEST AND EAST BOUNDARIES */

			int NJ2 = 2 * NJ;

			for (int J = 1; J < NJM; ++J)
			{
				int IJ = LI[0] + J;
				P[IJ] = P[IJ + NJ] + (P[IJ + NJ] - P[IJ + NJ2])*FX[1];
				IJ = LI[NI - 1] + J;
				P[IJ] = P[IJ - NJ] + (P[IJ - NJ] - P[IJ - NJ2])*(1.0 - FX[NIM - 1 - 1]);
			}

			/*--------------------------------------------------------------------------------*/

			/* INITIALIZE TEMPORARILY STORED VARIABLES */

			for (int IJ = 0; IJ < NIJ; ++IJ)
			{
				SU[IJ] = 0.0;
				SV[IJ] = 0.0;
				APU[IJ] = 0.0;
				APV[IJ] = 0.0;
			}

			/* FLUXES THROUGH INTERNAL EAST CV FACES */

			/* F1(IJ) is the mass flux through the east face (outward normal directed to E);
			FX(I) is the ratio of distance from P to cell face, to distance from P to E; IJ
			denotes node P and IJE node E. Contribution of convective and diffusive fluxes
			from east face to AE(P), AW(E), and source terms at both P and E are calculated;
			contributions to AP(P) and AP(E) come through the sum of neighbor coefficients
			and are not explicitly calculated.*/

			for (int I = 1; I < NIM - 1; ++I)
			{
				/* INTERPOLATION FACTORS, DISTANCE FROM P TO E (SAME FOR ALL J) */

				FXE = FX[I];
				FXP = 1.0 - FXE;
				DXPE = XC[I + 1] - XC[I];

				for (int J = 1; J < NJM; ++J)
				{
					int IJ = LI[I] + J;
					int IJE = IJ + NJ;

					/* CELL FACE AREA S = DY*RE*1 */

					S = (Y[J] - Y[J - 1])*(R[J] + R[J - 1])*0.5;

					/* COEFFICIENT RESULTING FROM DIFFUSIVE FLUX (SAME FOR U AND V) */

					D = VIS*S / DXPE;

					/* EXPLICIT CONVECTIVE FLUXES FOR UDS AND CDS */

					CE = min(F1[IJ], 0.0);
					CP = max(F1[IJ], 0.0);

					FUUDS = CP*U[IJ] + CE*U[IJE];
					FVUDS = CP*V[IJ] + CE*V[IJE];
					FUCDS = F1[IJ] * (U[IJE] * FXE + U[IJ] * FXP);
					FVCDS = F1[IJ] * (V[IJE] * FXE + V[IJ] * FXP);

					/* COEFFICIENTS AE(P) AND AW(E) DUE TO UDS */

					AE[IJ] = CE - D;
					AW[IJE] = -CP - D;

					/* SOURCE TERM CONTRIBUTIONS AT P AND E DUE TO DEFERRED CORRECTION */

					SU[IJ] = SU[IJ] + GDSU*(FUUDS - FUCDS);
					SU[IJE] = SU[IJE] - GDSU*(FUUDS - FUCDS);
					SV[IJ] = SV[IJ] + GDSU*(FVUDS - FVCDS);
					SV[IJE] = SV[IJE] - GDSU*(FVUDS - FVCDS);
				}
			}

			/* FLUXES THROUGH INTERNAL NORTH CV FACES */

			/* F2(IJ) is the mass flux through the north face (outward normal directed to N);
			FY(J) is the ratio of distance from P to cell face, to distance from P to N;
			IJ denotes node P and IJN node N.Contribution of convective and diffusive fluxes
			from north face to AN(P), AS(N), and source terms at both P and N are calculated;
			contributions to AP(P) and AP(N) come through the sum of neighbor coefficients
			and are not explicitly calculated.*/

			for (int J = 1; J < NJM - 1; ++J)
			{
				/* INTERPOLATION FACTORS, DISTANCE FROM P TO N (SAME FOR ALL J) */

				FYN = FY[J];
				FYP = 1.0 - FYN;
				DYPN = YC[J + 1] - YC[J];

				for (int I = 1; I < NIM; ++I)
				{
					int IJ = LI[I] + J;
					int IJN = IJ + 1;

					/* CELL FACE AREA S = DX*RN*1 */

					S = (X[I] - X[I - 1])*R[J];

					/* COEFFICIENT RESULTING FROM DIFFUSIVE FLUX (SAME FOR U AND V) */

					D = VIS*S / DYPN;

					/* EXPLICIT CONVECTIVE FLUXES FOR UDS AND CDS */

					CN = min(F2[IJ], 0.0);
					CP = max(F2[IJ], 0.0);

					FVUDS = CP*V[IJ] + CN*V[IJN];
					FUUDS = CP*U[IJ] + CN*U[IJN];
					FUCDS = F2[IJ] * (U[IJN] * FYN + U[IJ] * FYP);
					FVCDS = F2[IJ] * (V[IJN] * FYN + V[IJ] * FYP);

					/* COEFFICIENTS AN(P) AND AS(N) DUE TO UDS */

					AN[IJ] = CN - D;
					AS[IJN] = -CP - D;

					/* SOURCE TERM CONTRIBUTIONS AT P AND N DUE TO DEFERRED CORRECTION */

					SU[IJ] = SU[IJ] + GDSU*(FUUDS - FUCDS);
					SU[IJN] = SU[IJN] - GDSU*(FUUDS - FUCDS);
					SV[IJ] = SV[IJ] + GDSU*(FVUDS - FVCDS);
					SV[IJN] = SV[IJN] - GDSU*(FVUDS - FVCDS);
				}
			}

			/* VOLUME INTEGRALS (SOURCE TERMS) */

			/* Cell-face pressure calculated using linear interpolation;
			cell volume is VOL, RP is the radius at node P; DX and DY
			are the width and height of the cell. Contribution to AP
			coefficient from volume integrals is stored temporarily
			in arrays APU and APV for U and V, respectively; these
			arrays are later used to store 1/AP, which is needed in
			the pressure-correction equation. */

			for (int I = 1; I < NIM; ++I)
			{
				DX = X[I] - X[I - 1];

				for (int J = 1; J < NJM; ++J)
				{
					DY = Y[J] - Y[J - 1];
					RP = 0.5*(R[J] + R[J - 1]);
					VOL = DX*DY*RP;
					int IJ = LI[I] + J;

					/* CELL-FACE PRESSURE, CELL-CENTER GRADIENT, SOURCE */

					PE = P[IJ + NJ] * FX[I] + P[IJ] * (1.0 - FX[I]);
					PW = P[IJ] * FX[I - 1] + P[IJ - NJ] * (1.0 - FX[I - 1]);
					PN = P[IJ + 1] * FY[J] + P[IJ] * (1.0 - FY[J]);
					PS = P[IJ] * FY[J - 1] + P[IJ - 1] * (1.0 - FY[J - 1]);
					DPX[IJ] = (PE - PW) / DX;
					DPY[IJ] = (PN - PS) / DY;
					SU[IJ] = SU[IJ] - DPX[IJ] * VOL;
					SV[IJ] = SV[IJ] - DPY[IJ] * VOL;

					/* BUOYANCY SOURCE CONTRIBUTION */

					if (LCALIEN)
					{
						SB = -BETA*DEN*VOL*(T[IJ] - TREF);
						SU[IJ] = SU[IJ] + GRAVX*SB;
						SV[IJ] = SV[IJ] + GRAVY*SB;
					}

					/* AXISYMMETRIC CONTRIBUTION */

					//IF(LAXIS) THEN
					//APV(IJ) = APV(IJ) + VISC*VOL / RP**2
					//ENDIF

					/* UNSTEADY TERM CONTRIBUTION TO AP AND SU */

					if (LTIME)
					{
						APT = DEN*VOL*DTR;
						SU[IJ] = SU[IJ] + (1.0 + GAMT)*APT*UO[IJ] - 0.5*GAMT*APT*UOO[IJ];
						SV[IJ] = SV[IJ] + (1.0 + GAMT)*APT*VO[IJ] - 0.5*GAMT*APT*VOO[IJ];
						APV[IJ] = APV[IJ] + (1.0 + 0.5*GAMT)*APT;
						APU[IJ] = APU[IJ] + (1.0 + 0.5*GAMT)*APT;
					}
				}
			}

			/* PROBLEM MODIFICATIONS - BOUNDARY CONDITIONS */

			/*******************************************************************************/

			/* BCUV */

			/*  In this routine, boundary conditions for U and V equations
			are implemented, i.e. fluxes through boundary cell faces
			are approximated. Here, the boundaries encountered in
			cavity flows are considered; inlet and outlet boundaries
			require different treatment*/

			/* SOUTH BOUNDARY (WALL; SHEAR FORCE IN X-DIR, DV/DY=0) */

			for (int I = 1; I < NIM; ++I)
			{
				int IJ = LI[I] + 1;
				D = VIS*(X[I] - X[I - 1])*R[0] / (YC[1] - YC[0]);
				APU[IJ] = APU[IJ] + D;
				SU[IJ] = SU[IJ] + D*U[IJ - 1];
			}

			/* NORTH BOUNDARY (WALL, SHEAR FORCE IN X-DIR, DV/DY=0)*/

			for (int I = 1; I < NIM; ++I)
			{
				int IJ = LI[I] + NJM - 1;
				D = VIS*(X[I] - X[I - 1])*R[NJM - 1] / (YC[NJ - 1] - YC[NJM - 1]);
				APU[IJ] = APU[IJ] + D;
				SU[IJ] = SU[IJ] + D*U[IJ + 1];
			}

			/* WEST BOUNDARY (WALL, SHEAR FORCE IN Y-DIR, DU/DX=0) */

			for (int J = 1; J < NJM; ++J)
			{
				int IJ = LI[1] + J;
				D = 0.5*VIS*(Y[J] - Y[J - 1])*(R[J] + R[J - 1]) / (XC[1] - XC[0]);
				APV[IJ] = APV[IJ] + D;
				SV[IJ] = SV[IJ] + D*V[IJ - NJ];
			}

			/* EAST BOUNDARY (WALL, SHEAR FORCE IN Y-DIR, DU/DX=0) */

			for (int J = 1; J < NJM; ++J)
			{
				int IJ = LI[NIM - 1] + J;
				D = 0.5*VIS*(Y[J] - Y[J - 1])*(R[J] + R[J - 1]) / (XC[NI - 1] - XC[NIM - 1]);
				APV[IJ] = APV[IJ] + D;
				SV[IJ] = SV[IJ] + D*V[IJ + NJ];
			}

			/*******************************************************************************/

			/* UNDER-RELAXATION, SOLVING EQUATION SYSTEM FOR U-VELOCITY */

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					AP[IJ] = (-AE[IJ] - AW[IJ] - AN[IJ] - AS[IJ] + APU[IJ])*URFUR;
					SU[IJ] = SU[IJ] + (1.0 - URFU)*AP[IJ] * U[IJ];
					APU[IJ] = 1.0 / AP[IJ];
				}
			}

			/*******************************************************************************/

			/****************************************************/

			QueryPerformanceFrequency(&Frequency);
			QueryPerformanceCounter(&StartingTimeU);

			/***************************************************/

			////////////
			// SIPSOL //
			////////////



			/* COEFFICIENTS OF UPPER AND LOWER TRIANGULAR MATRICES */

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					LW[IJ] = AW[IJ] / (1.0 + ALFA*UN[IJ - NJ]);
					LS[IJ] = AS[IJ] / (1.0 + ALFA*UE[IJ - 1]);
					P1 = ALFA*LW[IJ] * UN[IJ - NJ];
					P2 = ALFA*LS[IJ] * UE[IJ - 1];
					LPR[IJ] = 1.0 / (AP[IJ] + P1 + P2 - LW[IJ] * UE[IJ - NJ] - LS[IJ] * UN[IJ - 1]);
					UN[IJ] = (AN[IJ] - P1)*LPR[IJ];
					UE[IJ] = (AE[IJ] - P2)*LPR[IJ];
				}
			}

			/****************************************************/
			QueryPerformanceCounter(&StartingTimeTU);
			/***************************************************/
			hipMemcpy(D_LS, LS, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_LW, LW, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_UN, UN, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_UE, UE, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_LPR, LPR, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_U, U, NIJ*sizeof(double), hipMemcpyHostToDevice);
			/*******************************************************************************/
			QueryPerformanceCounter(&EndingTimeTU);
			ElapsedMicrosecondsTU.QuadPart = EndingTimeTU.QuadPart - StartingTimeTU.QuadPart;
			ElapsedMicrosecondsTU.QuadPart *= 1000000;
			ElapsedMicrosecondsTU.QuadPart /= Frequency.QuadPart;
			TIMETU += ElapsedMicrosecondsTU.QuadPart;
			/*********************************************************************************/


			/* INNER ITERATIONS LOOP*/

			for (int L = 0; L < MAXITU; ++L)
			{
				RESL = 0.0;

				/* CALCULATE RESIDUAL AND OVERWRITE IT BY INTERMEDIATE VECTOR */

				for (int I = 1; I < NIM; ++I)
				{
					int Start = (LI[I] + 1);
					int End = (LI[I] + NJM);

					for (int IJ = Start; IJ < End; ++IJ)
					{
						RES[IJ] = SU[IJ] - AN[IJ] * U[IJ + 1] - AS[IJ] * U[IJ - 1] -
							AE[IJ] * U[IJ + NJ] - AW[IJ] * U[IJ - NJ] - AP[IJ] * U[IJ];

						double ABS = abs(RES[IJ]);
						RESL += ABS;
					}
				}


				hipMemcpy(D_RES, RES, NIJ*sizeof(double), hipMemcpyHostToDevice);


				Forward << <1, 1 >> >(D_RES, D_LS, D_LW, D_LPR, NI, NJ, thrds);

				/* STORE INITIAL RESIDUAL SUM FOR CHECKING CONV. OF OUTER ITER. */

				if (L == 0)
				{
					RESOR = RESL;
				}

				RSM = RESL / (RESOR + SMALL);

				/* BACK SUBSTITUTION AND CORRECTION */

				Backward << <1, 1 >> >(D_U, D_RES, D_UN, D_UE, D_LPR, NI, NJ, thrds);


				hipMemcpy(U, D_U, NIJ*sizeof(double), hipMemcpyDeviceToHost);


				RSMU = RESOR;

				if (RSM < RESMAXU)
				{
					L = MAXITU;
				}

			}



			/*******************************************************************************/
			QueryPerformanceCounter(&EndingTimeU);
			ElapsedMicrosecondsU.QuadPart = EndingTimeU.QuadPart - StartingTimeU.QuadPart;
			ElapsedMicrosecondsU.QuadPart *= 1000000;
			ElapsedMicrosecondsU.QuadPart /= Frequency.QuadPart;
			TIMEU += ElapsedMicrosecondsU.QuadPart;
			/*********************************************************************************/

			/* UNDER-RELAXATION, SOLVING EQUATION SYSTEM FOR V-VELOCITY */

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					AP[IJ] = (-AE[IJ] - AW[IJ] - AN[IJ] - AS[IJ] + APV[IJ])*URFVR;
					SU[IJ] = SV[IJ] + (1.0 - URFV)*AP[IJ] * V[IJ];
					APV[IJ] = 1.0 / AP[IJ];
				}
			}

			/*******************************************************************************/

			/****************************************************/

			QueryPerformanceFrequency(&Frequency);
			QueryPerformanceCounter(&StartingTimeV);

			/***************************************************/


			////////////
			// SIPSOL //
			////////////

			/* COEFFICIENTS OF UPPER AND LOWER TRIANGULAR MATRICES */

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					LW[IJ] = AW[IJ] / (1.0 + ALFA*UN[IJ - NJ]);
					LS[IJ] = AS[IJ] / (1.0 + ALFA*UE[IJ - 1]);
					P1 = ALFA*LW[IJ] * UN[IJ - NJ];
					P2 = ALFA*LS[IJ] * UE[IJ - 1];
					LPR[IJ] = 1.0 / (AP[IJ] + P1 + P2 - LW[IJ] * UE[IJ - NJ] - LS[IJ] * UN[IJ - 1]);
					UN[IJ] = (AN[IJ] - P1)*LPR[IJ];
					UE[IJ] = (AE[IJ] - P2)*LPR[IJ];
				}
			}

			hipMemcpy(D_LS, LS, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_LW, LW, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_UN, UN, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_UE, UE, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_LPR, LPR, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_V, V, NIJ*sizeof(double), hipMemcpyHostToDevice);


			/* INNER ITERATIONS LOOP*/

			for (int L = 0; L < MAXITV; ++L)
			{
				RESL = 0.0;

				/* CALCULATE RESIDUAL AND OVERWRITE IT BY INTERMEDIATE VECTOR */

				for (int I = 1; I < NIM; ++I)
				{
					int Start = (LI[I] + 1);
					int End = (LI[I] + NJM);

					for (int IJ = Start; IJ < End; ++IJ)
					{
						RES[IJ] = SU[IJ] - AN[IJ] * V[IJ + 1] - AS[IJ] * V[IJ - 1] -
							AE[IJ] * V[IJ + NJ] - AW[IJ] * V[IJ - NJ] - AP[IJ] * V[IJ];

						double ABS = abs(RES[IJ]);
						RESL += ABS;
					}
				}

				hipMemcpy(D_RES, RES, NIJ*sizeof(double), hipMemcpyHostToDevice);

				int threads;

				/*for (int J = 2; J <= (NI + NI - 2); ++J)
				{
				int Start = max(1, J - NI + 2);
				int End = min(NI - 2, J - 1);

				threads = End - Start + 1;
				dim3 block(thrds);
				dim3 grid((threads + block.x - 1) / block.x);

				if ((End - Start + 1) > 0)
				{
				Forwardsub << <grid, block >> >(D_RES, D_LS, D_LW, D_LPR, NI, NJ, Start, J, threads);
				hipDeviceSynchronize();
				}
				}*/

				Forward << <1, 1 >> >(D_RES, D_LS, D_LW, D_LPR, NI, NJ, thrds);


				/* STORE INITIAL RESIDUAL SUM FOR CHECKING CONV. OF OUTER ITER. */

				if (L == 0)
				{
					RESOR = RESL;
				}

				RSM = RESL / (RESOR + SMALL);

				/* BACK SUBSTITUTION AND CORRECTION */

				/*for (int J = (NI + NI - 2); J >= 2; --J)
				{
				int Start = max(1, J - NI + 2);
				int End = min(NI - 2, J - 1);

				threads = End - Start + 1;
				dim3 block(thrds);
				dim3 grid((threads + block.x - 1) / block.x);


				if ((End - Start + 1) > 0)
				{
				Backwardsub << <grid, block >> >(D_V, D_RES, D_UN, D_UE, D_LPR, NI, NJ, End, J, threads);
				hipDeviceSynchronize();
				}
				}*/

				Backward << <1, 1 >> >(D_V, D_RES, D_UN, D_UE, D_LPR, NI, NJ, thrds);

				hipMemcpy(V, D_V, NIJ*sizeof(double), hipMemcpyDeviceToHost);

				/*for (int I = (NIM - 1); I > 0; --I)
				{
				int Start = (LI[I] + NJM - 1);
				int End = (LI[I] + 1);

				for (int IJ = Start; IJ >= End; --IJ)
				{
				V[IJ] = V[IJ] + RES[IJ];
				}
				}*/

				RSMV = RESOR;

				if (RSM < RESMAXV)
				{
					L = MAXITV;
				}

			}

			/*******************************************************************************/



			QueryPerformanceCounter(&EndingTimeV);
			ElapsedMicrosecondsV.QuadPart = EndingTimeV.QuadPart - StartingTimeV.QuadPart;
			ElapsedMicrosecondsV.QuadPart *= 1000000;
			ElapsedMicrosecondsV.QuadPart /= Frequency.QuadPart;
			TIMEV += ElapsedMicrosecondsV.QuadPart;

			/*********************************************************************************/

			///////////
			// CALCP //
			///////////



			/* This routine assembles and solves the pressure-correction
			equation. Cell-face values of velocity components, used to
			calculate the mass fluxes, are obtained by linear interpolation
			and then corrected by adding a term proportional to the third derivative
			of pressure and squared grid spacing, */

			/* EAST CV FACES (S - AREA, VOLE - VOLUME BETWEEN P AND E) */

			for (int I = 1; I < NIM - 1; ++I)
			{
				DXPE = XC[I + 1] - XC[I];
				FXE = FX[I];
				FXP = 1.0 - FXE;

				for (int J = 1; J < NJM; ++J)
				{
					int IJ = LI[I] + J;
					int	IJE = IJ + NJ;

					S = (Y[J] - Y[J - 1])*(R[J] + R[J - 1])*0.5;
					VOLE = DXPE*S;
					D = DEN*S;

					/* INTERPOLATED CELL FACE QUANTITIES (PRESSURE GRAD., U AND 1/AP)
					Note: pressure gradient is interpolated midway between P and E,
					since the gradient calculated at cell face is second order
					accurate at that location; the velocity is interpolated linearly,
					to achieve second order accuracy at cell face center. */

					DPXEL = 0.5*(DPX[IJE] + DPX[IJ]);
					UEL = U[IJE] * FXE + U[IJ] * FXP;
					APUE = APU[IJE] * FXE + APU[IJ] * FXP;

					/* CELL FACE GRADIENT, VELOCITY AND MASS FLUX */

					DPXE = (P[IJE] - P[IJ]) / DXPE;
					UEP = UEL - APUE*VOLE*(DPXE - DPXEL);
					F1[IJ] = D*UEP;

					/* COEFFICIENTS OF P' EQUATION, AE(P) AND AW(E) */

					AE[IJ] = -D*APUE*S;
					AW[IJE] = AE[IJ];
				}
			}

			/* NORTH CV FACES (S - AREA, VOLN - VOLUME BETWEEN P AND N) */

			for (int J = 1; J < NJM - 1; ++J)
			{
				DYPN = YC[J + 1] - YC[J];
				FYN = FY[J];
				FYP = 1.0 - FYN;

				for (int I = 1; I < NIM; ++I)
				{
					int IJ = LI[I] + J;
					int IJN = IJ + 1;

					S = (X[I] - X[I - 1])*R[J];
					VOLN = S*DYPN;
					D = DEN*S;

					/* INTERPOLATED CELL-FACE QUANTITIES (PRESSURE GRAD., U AND 1/AP) */

					DPYNL = 0.5*(DPY[IJN] + DPY[IJ]);
					VNL = V[IJN] * FYN + V[IJ] * FYP;
					APVN = APV[IJN] * FYN + APV[IJ] * FYP;

					/* CELL-FACE GRADIENT, VELOCITY AND MASS FLUX */

					DPYN = (P[IJN] - P[IJ]) / DYPN;
					VN = VNL - APVN*VOLN*(DPYN - DPYNL);
					F2[IJ] = D*VN;

					/* COEFFICIENTS OF P' EQUATION, AN(P) AND AS(N) */

					AN[IJ] = -D*APVN*S;
					AS[IJN] = AN[IJ];
				}
			}

			/* BOUNDARY CONDITIONS: PRESCRIBED MASS FLUXES, ZERO CORRECTION
			(EQUIVALENT TO ZERO NORMAL GRADIENT FOR P'; COEFFICIENT FOR
			THE BOUNDARY NODE IS ZERO, NO SPECIAL TREATMENT REQUIRED)*/

			/* SORCE TERM AND COEFFICIENT OF NODE P*/

			SUM = 0.0;

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					SU[IJ] = F1[IJ - NJ] - F1[IJ] + F2[IJ - 1] - F2[IJ];
					AP[IJ] = -(AE[IJ] + AW[IJ] + AN[IJ] + AS[IJ]);
					SUM = SUM + SU[IJ];
					PP[IJ] = 0.0;
				}
			}

			/* SUM MUST BE ZERO IF GLOBAL MASS CONSERVATION IS ASSURED! */

			/* IF(LTEST) WRITE(2,*) '       SUM = ',SUM */

			/* SOLVE EQUATIONS SYSTEM FOR P' AND APPLY CORRECTIONS */

			/*******************************************************************************/

			/****************************************************/

			QueryPerformanceFrequency(&Frequency);
			QueryPerformanceCounter(&StartingTimeP);

			/***************************************************/

			////////////
			// SIPSOL //
			////////////

			/* COEFFICIENTS OF UPPER AND LOWER TRIANGULAR MATRICES */

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					LW[IJ] = AW[IJ] / (1.0 + ALFA*UN[IJ - NJ]);
					LS[IJ] = AS[IJ] / (1.0 + ALFA*UE[IJ - 1]);
					P1 = ALFA*LW[IJ] * UN[IJ - NJ];
					P2 = ALFA*LS[IJ] * UE[IJ - 1];
					LPR[IJ] = 1.0 / (AP[IJ] + P1 + P2 - LW[IJ] * UE[IJ - NJ] - LS[IJ] * UN[IJ - 1]);
					UN[IJ] = (AN[IJ] - P1)*LPR[IJ];
					UE[IJ] = (AE[IJ] - P2)*LPR[IJ];
				}
			}

			hipMemcpy(D_LS, LS, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_LW, LW, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_UN, UN, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_UE, UE, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_LPR, LPR, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_PP, PP, NIJ*sizeof(double), hipMemcpyHostToDevice);

			/* INNER ITERATIONS LOOP*/

			for (int L = 0; L < MAXITP; ++L)
			{
				RESL = 0.0;

				/* CALCULATE RESIDUAL AND OVERWRITE IT BY INTERMEDIATE VECTOR */

				for (int I = 1; I < NIM; ++I)
				{
					int Start = (LI[I] + 1);
					int End = (LI[I] + NJM);

					for (int IJ = Start; IJ < End; ++IJ)
					{
						RES[IJ] = SU[IJ] - AN[IJ] * PP[IJ + 1] - AS[IJ] * PP[IJ - 1] -
							AE[IJ] * PP[IJ + NJ] - AW[IJ] * PP[IJ - NJ] - AP[IJ] * PP[IJ];
						double ABS = abs(RES[IJ]);
						RESL += ABS;
					}
				}

				hipMemcpy(D_RES, RES, NIJ*sizeof(double), hipMemcpyHostToDevice);

				int threads;

				/*for (int J = 2; J <= (NI + NI - 2); ++J)
				{
				int Start = max(1, J - NI + 2);
				int End = min(NI - 2, J - 1);

				threads = End - Start + 1;
				dim3 block(thrds);
				dim3 grid((threads + block.x - 1) / block.x);

				if ((End - Start + 1) > 0)
				{
				Forwardsub << <grid, block >> >(D_RES, D_LS, D_LW, D_LPR, NI, NJ, Start, J, threads);
				hipDeviceSynchronize();
				}
				}*/

				Forward << <1, 1 >> >(D_RES, D_LS, D_LW, D_LPR, NI, NJ, thrds);

				/* STORE INITIAL RESIDUAL SUM FOR CHECKING CONV. OF OUTER ITER. */

				if (L == 0)
				{
					RESOR = RESL;
				}

				RSM = RESL / (RESOR + SMALL);

				/* BACK SUBSTITUTION AND CORRECTION */

				/*for (int J = (NI + NI - 2); J >= 2; --J)
				{
				int Start = max(1, J - NI + 2);
				int End = min(NI - 2, J - 1);

				threads = End - Start + 1;
				dim3 block(thrds);
				dim3 grid((threads + block.x - 1) / block.x);

				if ((End - Start + 1) > 0)
				{
				Backwardsub << <grid, block >> >(D_PP, D_RES, D_UN, D_UE, D_LPR, NI, NJ, End, J, threads);
				hipDeviceSynchronize();
				}
				}*/

				Backward << <1, 1 >> >(D_PP, D_RES, D_UN, D_UE, D_LPR, NI, NJ, thrds);

				hipMemcpy(PP, D_PP, NIJ*sizeof(double), hipMemcpyDeviceToHost);

				/*for (int I = (NIM - 1); I > 0; --I)
				{
				int Start = (LI[I] + NJM - 1);
				int End = (LI[I] + 1);

				for (int IJ = Start; IJ >= End; --IJ)
				{
				PP[IJ] = PP[IJ] + RES[IJ];
				}
				}*/

				RSMP = RESOR;

				if (RSM < RESMAXP)
				{
					L = MAXITP;
				}
			}

			/*********************************************************************************/

			QueryPerformanceCounter(&EndingTimeP);
			ElapsedMicrosecondsP.QuadPart = EndingTimeP.QuadPart - StartingTimeP.QuadPart;
			ElapsedMicrosecondsP.QuadPart *= 1000000;
			ElapsedMicrosecondsP.QuadPart /= Frequency.QuadPart;
			TIMEP += ElapsedMicrosecondsP.QuadPart;

			/*********************************************************************************/

			/*******************************************************************************/

			/* CALCULATE PRESSURE CORRECTION AT BOUNDARIES */

			/*--------------------------------------------------------------------------------*/

			/* PBOUND */

			/* This routine calculates boundary values of pressure or
			pressure-correction by extrapolating (linearly) from inside.*/

			/* SOUTH AND NORTH BOUNDARIES */

			for (int I = 1; I < NIM; ++I)
			{
				int IJ = LI[I];
				PP[IJ] = PP[IJ + 1] + (PP[IJ + 1] - PP[IJ + 2])*FY[1];
				IJ = LI[I] + NJ - 1;
				PP[IJ] = PP[IJ - 1] + (PP[IJ - 1] - PP[IJ - 2])*(1.0 - FY[NJM - 1 - 1]);
			}

			/* WEST AND EAST BOUNDARIES */

			NJ2 = 2 * NJ;

			for (int J = 1; J < NJM; ++J)
			{
				int IJ = LI[0] + J;
				PP[IJ] = PP[IJ + NJ] + (PP[IJ + NJ] - PP[IJ + NJ2])*FX[1];
				IJ = LI[NI - 1] + J;
				PP[IJ] = PP[IJ - NJ] + (PP[IJ - NJ] - PP[IJ - NJ2])*(1.0 - FX[NIM - 1 - 1]);
			}

			/*--------------------------------------------------------------------------------*/

			/* VALUE OF P' AT REFERENCE LOCATION TO BE SUBTRACTED FROM ALL P' */

			IJPREF = LI[IPR] + JPR;
			PPO = PP[IJPREF];

			/* CORRECT EAST MASS FLUXES  */

			for (int I = 1; I < NIM - 1; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					F1[IJ] = F1[IJ] + AE[IJ] * (PP[IJ + NJ] - PP[IJ]);
				}
			}

			/* CORRECT NORTH MASS FLUXES */

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NIM - 1); ++IJ)
				{
					F2[IJ] = F2[IJ] + AN[IJ] * (PP[IJ + 1] - PP[IJ]);
				}
			}

			/* CORRECT PRESSURE AND VELOCITIES AT CELL CENTER */

			for (int I = 1; I < NIM; ++I)
			{
				DX = X[I] - X[I - 1];

				for (int J = 1; J < NJM; ++J)
				{
					int IJ = LI[I] + J;
					RP = 0.5*(R[J] + R[J - 1]);
					DY = Y[J] - Y[J - 1];

					PPE = PP[IJ + NJ] * FX[I] + PP[IJ] * (1.0 - FX[I]);
					PPW = PP[IJ] * FX[I - 1] + PP[IJ - NJ] * (1.0 - FX[I - 1]);
					PPN = PP[IJ + 1] * FY[J] + PP[IJ] * (1.0 - FY[J]);
					PPS = PP[IJ] * FY[J - 1] + PP[IJ - 1] * (1.0 - FY[J - 1]);

					U[IJ] = U[IJ] - (PPE - PPW)*DY*RP*APU[IJ];
					V[IJ] = V[IJ] - (PPN - PPS)*DX*RP*APV[IJ];
					P[IJ] = P[IJ] + URFP*(PP[IJ] - PPO);
				}
			}


			///////////
			// CALCT //
			///////////



			/* This routine solves the temperature equation. Constant
			viscosity, density and Prandtl number are assumed - only
			the density variation due to buoyancy is considered
			using Boussinesq approximation (valid for temperature
			differences less than 5 deg. in water and 20 deg. in air).
			PRR is the reciprocal value of the Prandtl number, 1/Pr.*/

			/* INITIALIZATION OF TEMPORARILY STORED VARIABLES */

			for (int IJ = 0; IJ < NIJ; ++IJ)
			{
				SU[IJ] = 0.0;
				AP[IJ] = 0.0;
			}

			URFTR = 1.0 / URFT;

			/* FLUXES THROUGH INTERNAL EAST CV-FACES */

			for (int I = 1; I < NIM - 1; ++I)
			{
				/* INTERPOLATION FACTORS, DISTANCE FROM P TO E (SAME FOR ALL J)*/

				FXE = FX[I];
				FXP = 1.0 - FXE;
				DXPE = XC[I + 1] - XC[I];

				for (int J = 1; J < NJM; ++J)
				{
					int IJ = LI[I] + J;
					int	IJE = IJ + NJ;

					/* CELL FACE AREA S = DY*RE*1 */

					S = (Y[J] - Y[J - 1])*(R[J] + R[J - 1])*0.5;

					/* COEFFICIENT RESULTING FROM DIFFUSIVE FLUX */

					D = VIS*PRR*S / DXPE;

					/* EXPLICIT CONVECTIVE FLUX FOR UDS AND CDS */

					CE = min(F1[IJ], 0.0);
					CP = max(F1[IJ], 0.0);

					FUDS = CP*T[IJ] + CE*T[IJE];
					FCDS = F1[IJ] * (T[IJE] * FXE + T[IJ] * FXP);

					/* COEFFICIENTS AE(P) AND AW(E) DUE TO UDS */

					AE[IJ] = CE - D;
					AW[IJE] = -CP - D;

					/* SOURCE TERM CONTRIBUTIONS AT P AND E DUE TO DEFERRED CORRECTION */

					SU[IJ] = SU[IJ] + GDST*(FUDS - FCDS);
					SU[IJE] = SU[IJE] - GDST*(FUDS - FCDS);
				}
			}

			/* FLUXES THROUGH INTERNAL NORTH CV FACES */

			for (int J = 1; J < NJM - 1; ++J)
			{
				/* INTERPOLATION FACTORS, DISTANCE FROM P TO N (SAME FOR ALL J) */

				FYN = FY[J];
				FYP = 1.0 - FYN;
				DYPN = YC[J + 1] - YC[J];

				for (int I = 1; I < NIM; ++I)
				{
					int IJ = LI[I] + J;
					int	IJN = IJ + 1;

					/* CELL FACE AREA S = DX*RN*1 */

					S = (X[I] - X[I - 1])*R[J];

					/* COEFFICIENT RESULTING FROM DIFFUSIVE FLUX (SAME FOR U AND V) */

					D = VIS*PRR*S / DYPN;

					/* EXPLICIT CONVECTIVE FLUXES FOR UDS AND CDS */

					CN = min(F2[IJ], 0.);
					CP = max(F2[IJ], 0.);

					FUDS = CP*T[IJ] + CN*T[IJN];
					FCDS = F2[IJ] * (T[IJN] * FYN + T[IJ] * FYP);

					/* COEFFICIENTS AE(P) AND AW(E) DUE TO UDS */

					AN[IJ] = CN - D;
					AS[IJN] = -CP - D;

					/* SOURCE TERM CONTRIBUTIONS AT P AND E DUE TO DEFERRED CORRECTION */

					SU[IJ] = SU[IJ] + GDST*(FUDS - FCDS);
					SU[IJN] = SU[IJN] - GDST*(FUDS - FCDS);
				}
			}

			/* VOLUME INTEGRALS (SOURCE TERMS) */

			for (int I = 1; I < NIM; ++I)
			{
				DX = X[I] - X[I - 1];

				for (int J = 1; J < NJM; ++J)
				{
					int IJ = LI[I] + J;
					DY = Y[J] - Y[J - 1];
					RP = 0.5*(R[J] + R[J - 1]);
					VOL = DX*DY*RP;

					/* UNSTEADY TERM CONTRIBUTION TO AP AND SU */

					if (LTIME)
					{
						APT = DEN*VOL*DTR;
						SU[IJ] = SU[IJ] + (1. + GAMT)*APT*TO[IJ] - 0.5*GAMT*APT*TOO[IJ];
						AP[IJ] = AP[IJ] + (1. + 0.5*GAMT)*APT;
					}
				}
			}

			/*--------------------------------------------------------------------------------*/

			/* BCT */

			/*  In this routine, boundary conditions for the temperature
			equation are implemented, i.e. heat fluxes through the
			boundary cell faces are calculated. Here, specified wall
			temperature and adiabatic wall (zero heat flux) are considered;
			treatment at symmetry planes is the same as for an adiabatic
			wall, but inlet and outlet require different treatment*/

			/* SOUTH BOUNDARY (ADIABATIC WALL, DT/DY=0, ZERO FLUX) */

			for (int I = 1; I < NIM; ++I)
			{
				int IJ = LI[I];
				T[IJ] = T[IJ + 1];
			}

			/* NORTH BOUNDARY (ADIABATIC WALL, DT/DY=0, ZERO FLUX) */

			for (int I = 1; I < NIM; ++I)
			{
				int IJ = LI[I] + NJ - 1;
				T[IJ] = T[IJ - 1];
			}

			/* WEST BOUNDARY (ISOTHERMAL WALL, NON-ZERO DIFFUSIVE FLUX) */

			for (int J = 1; J < NJM; ++J)
			{
				int IJ = LI[1] + J;
				D = 0.5*VIS*PRR*(Y[J] - Y[J - 1])*(R[J] + R[J - 1]) / (XC[1] - XC[0]);
				AP[IJ] = AP[IJ] + D;
				SU[IJ] = SU[IJ] + D*T[IJ - NJ];
			}


			/* EAST BOUNDARY (ISOTHERMAL WALL) */

			for (int J = 1; J < NJM; ++J)
			{
				int IJ = LI[NIM - 1] + J;
				D = 0.5*VIS*PRR*(Y[J] - Y[J - 1])*(R[J] + R[J - 1]) / (XC[NI - 1] - XC[NIM - 1]);
				AP[IJ] = AP[IJ] + D;
				SU[IJ] = SU[IJ] + D*T[IJ + NJ];
			}

			/*--------------------------------------------------------------------------------*/

			/* UNDER-RELAXATION, SOLVING EQUATION SYSTEM FOR TEMPERATURE */

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					AP[IJ] = (AP[IJ] - AW[IJ] - AE[IJ] - AN[IJ] - AS[IJ])*URFTR;
					SU[IJ] = SU[IJ] + (1.0 - URFT)*AP[IJ] * T[IJ];
				}
			}

			/*******************************************************************************/

			/****************************************************/

			QueryPerformanceFrequency(&Frequency);
			QueryPerformanceCounter(&StartingTimeT);

			/***************************************************/

			////////////
			// SIPSOL //
			////////////

			/* COEFFICIENTS OF UPPER AND LOWER TRIANGULAR MATRICES */

			for (int I = 1; I < NIM; ++I)
			{
				for (int IJ = (LI[I] + 1); IJ < (LI[I] + NJM); ++IJ)
				{
					LW[IJ] = AW[IJ] / (1.0 + ALFA*UN[IJ - NJ]);
					LS[IJ] = AS[IJ] / (1.0 + ALFA*UE[IJ - 1]);
					P1 = ALFA*LW[IJ] * UN[IJ - NJ];
					P2 = ALFA*LS[IJ] * UE[IJ - 1];
					LPR[IJ] = 1.0 / (AP[IJ] + P1 + P2 - LW[IJ] * UE[IJ - NJ] - LS[IJ] * UN[IJ - 1]);
					UN[IJ] = (AN[IJ] - P1)*LPR[IJ];
					UE[IJ] = (AE[IJ] - P2)*LPR[IJ];
				}
			}

			hipMemcpy(D_LS, LS, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_LW, LW, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_UN, UN, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_UE, UE, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_LPR, LPR, NIJ*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(D_T, T, NIJ*sizeof(double), hipMemcpyHostToDevice);

			/* INNER ITERATIONS LOOP*/

			for (int L = 0; L < MAXITT; ++L)
			{
				RESL = 0.0;

				/* CALCULATE RESIDUAL AND OVERWRITE IT BY INTERMEDIATE VECTOR */

				for (int I = 1; I < NIM; ++I)
				{
					int Start = (LI[I] + 1);
					int End = (LI[I] + NJM);

					for (int IJ = Start; IJ < End; ++IJ)
					{
						RES[IJ] = SU[IJ] - AN[IJ] * T[IJ + 1] - AS[IJ] * T[IJ - 1] -
							AE[IJ] * T[IJ + NJ] - AW[IJ] * T[IJ - NJ] - AP[IJ] * T[IJ];
						double ABS = abs(RES[IJ]);
						RESL += ABS;
					}
				}

				hipMemcpy(D_RES, RES, NIJ*sizeof(double), hipMemcpyHostToDevice);

				int threads;

				/*for (int J = 2; J <= (NI + NI - 2); ++J)
				{
				int Start = max(1, J - NI + 2);
				int End = min(NI - 2, J - 1);

				threads = End - Start + 1;
				dim3 block(thrds);
				dim3 grid((threads + block.x - 1) / block.x);

				if ((End - Start + 1) > 0)
				{
				Forwardsub << <grid, block >> >(D_RES, D_LS, D_LW, D_LPR, NI, NJ, Start, J, threads);
				hipDeviceSynchronize();
				}
				}*/

				Forward << <1, 1 >> >(D_RES, D_LS, D_LW, D_LPR, NI, NJ, thrds);


				/* STORE INITIAL RESIDUAL SUM FOR CHECKING CONV. OF OUTER ITER. */

				if (L == 0)
				{
					RESOR = RESL;
				}

				RSM = RESL / (RESOR + SMALL);

				/* BACK SUBSTITUTION AND CORRECTION */

				/*for (int J = (NI + NI - 2); J >= 2; --J)
				{
				int Start = max(1, J - NI + 2);
				int End = min(NI - 2, J - 1);

				threads = End - Start + 1;
				dim3 block(thrds);
				dim3 grid((threads + block.x - 1) / block.x);

				if ((End - Start + 1) > 0)
				{
				Backwardsub << <grid, block >> >(D_T, D_RES, D_UN, D_UE, D_LPR, NI, NJ, End, J, threads);
				hipDeviceSynchronize();
				}
				}*/

				Backward << <1, 1 >> >(D_T, D_RES, D_UN, D_UE, D_LPR, NI, NJ, thrds);

				hipMemcpy(T, D_T, NIJ*sizeof(double), hipMemcpyDeviceToHost);

				/*for (int I = (NIM - 1); I > 0; --I)
				{
				int Start = (LI[I] + NJM - 1);
				int End = (LI[I] + 1);

				for (int IJ = Start; IJ >= End; --IJ)
				{
				T[IJ] = T[IJ] + RES[IJ];
				}
				}*/

				RSMT = RESOR;

				if (RSM < RESMAXT)
				{
					L = MAXITT;
				}

			}

			/*******************************************************************************/

			/*********************************************************************************/

			QueryPerformanceCounter(&EndingTimeT);
			ElapsedMicrosecondsT.QuadPart = EndingTimeT.QuadPart - StartingTimeT.QuadPart;
			ElapsedMicrosecondsT.QuadPart *= 1000000;
			ElapsedMicrosecondsT.QuadPart /= Frequency.QuadPart;
			TIMET += ElapsedMicrosecondsT.QuadPart;

			/*********************************************************************************/

			/////////////////////////////////////////////////////////////////////////////////////////////////////////



			outputfile << setw(5) << fixed << setprecision(3) << right << setfill('0') << ITER + 1 << "  |  "
				<< setw(5) << scientific << setprecision(3) << left << setfill('0') << RSMU << "  |  "
				<< setw(5) << scientific << setprecision(3) << left << setfill('0') << RSMV << "  |  "
				<< setw(5) << scientific << setprecision(3) << left << setfill('0') << RSMP << "  |  "
				<< setw(5) << scientific << setprecision(3) << left << setfill('0') << RSMT << "  ||  "
				<< setw(8) << scientific << setprecision(6) << left << setfill('0') << U[IJMON] << "  |  "
				<< setw(8) << scientific << setprecision(6) << left << setfill('0') << V[IJMON] << "  |  "
				<< setw(8) << scientific << setprecision(6) << left << setfill('0') << P[IJMON] << "  |  "
				<< setw(8) << scientific << setprecision(6) << left << setfill('0') << T[IJMON] << endl;


			////////////////////////////////////////////////////////////////////////////////////////////////////////
			//
			//			SOURCE = max(max(RSMU, RSMV), max(RSMP, RSMT));
			SOURCE = RSMU;
			if (SOURCE < RESMAX)
			{
				outputfile << endl << "Outer Iteration Converged after: " << ITER + 1 << " Iterations" << endl;
				ITER = MAXIT;
			}

			else if (SOURCE > SLARGE)
			{
				outputfile << endl << "Outer Iteration Diverged after: " << ITER + 1 << " Iterations" << endl;
				ITER = MAXIT;
			}

			/////////////////////////////////////////////////////////////////////////////////////////////////////

			/* Progress Bar*/
			int barWidth = 70;
			std::cout << "[";
			int pos = barWidth * (progress / MAXIT);
			for (int i = 0; i < barWidth; ++i)
			{
				if (i < pos) std::cout << "=";
				else if (i == pos) std::cout << ">";
				else std::cout << " ";
			}
			std::cout << "] " << int((progress * 100.0) / MAXIT) << " %\r";
			std::cout.flush();
			progress = ITER;

			///////////////////////////////////////////////////////////////////////////////////////////////////

			if (InterSave == "True" || InterSave == "true")
			{
				if ((ITER%InterSaveNo) == 0 && ITER != 0)
				{
					Intermediate.open(string(Ofilename + ".inter").c_str(), ios::out);
					Intermediate.seekp(0, Intermediate.beg);
					Intermediate << ITER << endl;
					for (int I = 1; I < NI; ++I)
					{
						LI[I] = I*NJ;
						for (int J = 1; J < NJ; ++J)
						{
							int IJ = LI[I] + J;
							Intermediate << setw(5) << fixed << setprecision(3) << right << setfill('0') << I << "     "
								<< setw(5) << fixed << setprecision(3) << right << setfill('0') << J << "     "
								<< setw(18) << fixed << setprecision(12) << left << setfill('0') << U[IJ] << "     "
								<< setw(18) << fixed << setprecision(12) << left << setfill('0') << V[IJ] << "     "
								<< setw(18) << fixed << setprecision(12) << left << setfill('0') << P[IJ] << "     "
								<< setw(18) << fixed << setprecision(12) << left << setfill('0') << T[IJ] << endl;
						}
					}
					Intermediate.close();
				}

			}

		}
	}

	/*****************************************************************************************************/

	QueryPerformanceCounter(&EndingTimeALL);
	ElapsedMicrosecondsALL.QuadPart = EndingTimeALL.QuadPart - StartingTimeALL.QuadPart;
	ElapsedMicrosecondsALL.QuadPart *= 1000000;
	ElapsedMicrosecondsALL.QuadPart /= Frequency.QuadPart;
	double Time = ElapsedMicrosecondsALL.QuadPart;

	/****************************************************************************************************/


	//////////////////////////////////////////////////////////////////////////////////////////////////////

	outputfile << endl << endl << "Elapsed Time solving U-Equation : "
		<< setw(10) << left << fixed << setprecision(6) << TIMEU / 1000000 << " Sec." << endl;
	outputfile << "Elapsed Time solving V-Equation : "
		<< setw(10) << left << fixed << setprecision(6) << TIMEV / 1000000 << " Sec." << endl;
	outputfile << "Elapsed Time solving P-Equation : "
		<< setw(10) << left << fixed << setprecision(6) << TIMEP / 1000000 << " Sec." << endl;
	outputfile << "Elapsed Time solving T-Equation : "
		<< setw(10) << left << fixed << setprecision(6) << TIMET / 1000000 << " Sec." << endl;

	outputfile << endl << endl;

	outputfile << "Percentage of Elapsed Time solving U-Equation : "
		<< setw(5) << left << fixed << setprecision(3) << (TIMEU / Time) * 100 << " % " << endl;
	outputfile << "Percentage of Elapsed Time solving V-Equation : "
		<< setw(5) << left << fixed << setprecision(3) << (TIMEV / Time) * 100 << " % " << endl;
	outputfile << "Percentage of Elapsed Time solving P-Equation : "
		<< setw(5) << left << fixed << setprecision(3) << (TIMEP / Time) * 100 << " % " << endl;
	outputfile << "Percentage of Elapsed Time solving T-Equation : "
		<< setw(5) << left << fixed << setprecision(3) << (TIMET / Time) * 100 << " % " << endl;

	outputfile << endl << endl;

	outputfile << "Percentage of Total Elapsed Time solving The Equations : "
		<< setw(5) << left << fixed << setprecision(3) << ((TIMEU + TIMEV + TIMEP + TIMET) / Time) * 100 << " % " << endl;

	outputfile << endl << endl;

	outputfile << "Total Elapsed Time : "
		<< setw(10) << left << fixed << setprecision(6) << Time / 1000000 << " Sec." << endl;

	//////////////////////////////////////////////////////////////////////////////////////////////////////

	///////////////////////////////////////////////////////////////////////////////////////////////////////////

	if (DAT == "True" || DAT == "true")
	{
		resultfile << "TITLE = \"PREDICTION OF FLOW IN LID DRIVEN CAVITIES\" " << endl;
		resultfile << "VARIABLES = \"X\", \"Y\", \"U\", \"V\", \"Press\", \"Temp\"" << endl;
		resultfile << "ZONE T=\"Cavity\"" << endl;
		resultfile << "I=" << NIM << ", " << "J=" << NJM << endl;
		resultfile << "ZONETYPE=Ordered" << endl;
		resultfile << "DATAPACKING=POINT" << endl;
		resultfile << "DT=(LONGINT LONGINT double double double double )" << endl << endl;

		for (int I = 1; I < NI; ++I)
		{
			LI[I] = I*NJ;
			for (int J = 1; J < NJ; ++J)
			{
				int IJ = LI[I] + J;
				resultfile << setw(5) << fixed << setprecision(3) << right << setfill('0') << I << "     "
					<< setw(5) << fixed << setprecision(3) << right << setfill('0') << J << "     "
					<< setw(18) << fixed << setprecision(12) << left << setfill('0') << U[IJ] << "     "
					<< setw(18) << fixed << setprecision(12) << left << setfill('0') << V[IJ] << "     "
					<< setw(18) << fixed << setprecision(12) << left << setfill('0') << P[IJ] << "     "
					<< setw(18) << fixed << setprecision(12) << left << setfill('0') << T[IJ] << endl;
			}
		}
	}


	//int IPSIMAX, JPSIMAX, IPSIMIN, JPSIMIN;

	///* STREAMFUNCTION VALUES AT CV-VERTICES (ZERO AT SOUTH-WEST CORNER) */

	//PP[LI[0] + 1] = 0.0;

	///* WEST BOUNDARY (APPLICABLE FOR INLET OR OUTLET) */

	//for (int J = 1; J < NJM; ++J)
	//{
	//	int IJ = LI[0] + J;
	//	PP[IJ] = PP[IJ - 1] - F1[IJ];
	//}

	///* SOUTH BOUNDARY (APPLICABLE FOR INLET OR OUTLET) */

	//for (int I = 1; I < NIM; ++I)
	//{
	//	int IJ = LI[I] + 1;
	//	PP[IJ] = PP[IJ] - F2[IJ];

	//	/* Inner Region */
	//	for (int J = 1; J < NJM; ++J)
	//	{
	//		int IJ = LI[I] + J;
	//		PP[IJ] = PP[IJ - 1] - F1[IJ];
	//	}
	//}

	///* STRENGTH OF PRIMARY AND SECONDARY EDDY (MIN and MAX values) */

	//double UIJ, VIJ, Unorth, Usouth, Veast, Vwest;
	//double OmegaMAX, OmegaMIN, DUDY, DVDX;
	//double PRIPSI, PRIOMEGA, PRII, PRIJ;
	//double BR1PSI, BR1OMEGA, BR1I, BR1J;
	//double BR2PSI, BR2OMEGA, BR2I, BR2J;
	//double BR3PSI, BR3OMEGA, BR3I, BR3J;
	//double BL1PSI, BL1OMEGA, BL1I, BL1J;
	//double BL2PSI, BL2OMEGA, BL2I, BL2J;
	//double BL3PSI, BL3OMEGA, BL3I, BL3J;
	//double TL1PSI, TL1OMEGA, TL1I, TL1J;
	//double TL2PSI, TL2OMEGA, TL2I, TL2J;

	//double PSIMAX = PRIPSI = BR2PSI = BL1PSI = TL1PSI = SMALL;
	//double PSIMIN = BR1PSI = BR3PSI = BL2PSI = TL2PSI = LARGE;

	///* Search for Psi, Omega Over the entire domain "Primary, and BR1" */

	//for (int I = 0; I < NIM; ++I)
	//{
	//	for (int J = 0; J < NJM; ++J)
	//	{
	//		int IJ = LI[I] + J;

	//		if (PP[IJ] < BR1PSI)
	//		{
	//			BR1PSI = PP[IJ];
	//			BR1I = I;
	//			BR1J = J;
	//			Unorth = U[IJ + 1];
	//			Usouth = U[IJ - 1];
	//			Veast = V[LI[I + 1] + J];
	//			Vwest = V[LI[I - 1] + J];
	//			DUDY = (Unorth - Usouth) * 0.5 * M;
	//			DVDX = (Veast - Vwest) * 0.5 * N;
	//			BR1OMEGA = DVDX - DUDY;
	//		}

	//		if (PP[IJ] > PRIPSI)
	//		{
	//			PRIPSI = PP[IJ];
	//			PRII = I;
	//			PRIJ = J;
	//			Unorth = U[IJ + 1];
	//			Usouth = U[IJ - 1];
	//			Veast = V[LI[I + 1] + J];
	//			Vwest = V[LI[I - 1] + J];
	//			DUDY = (Unorth - Usouth)*0.5*M;
	//			DVDX = (Veast - Vwest)*0.5*N;
	//			PRIOMEGA = DVDX - DUDY;
	//		}
	//	}
	//}

	//outputfile << endl << endl;
	//outputfile << "Strenght of The Primary Vortex : "
	//	<< setw(10) << left << fixed << setprecision(6) << PRIPSI << endl;
	//outputfile << "Location X: " << setw(8) << left << fixed << setprecision(6) << PRII*DY << endl;
	//outputfile << "Location Y: " << setw(8) << left << fixed << setprecision(6) << PRIJ*DX << endl;
	//outputfile << "OmegaMax: " << setw(8) << left << fixed << setprecision(6) << PRIOMEGA << endl;
	//outputfile << endl << endl;

	//outputfile << "Strenght of The BR1 Vortex : "
	//	<< setw(10) << left << fixed << setprecision(6) << BR1PSI << endl;
	//outputfile << "Location X: " << setw(8) << left << fixed << setprecision(6) << BR1I*DY << endl;
	//outputfile << "Location Y: " << setw(8) << left << fixed << setprecision(6) << BR1J*DX << endl;
	//outputfile << "OmegaMin: " << setw(8) << left << fixed << setprecision(6) << BR1OMEGA << endl;
	//outputfile << endl << endl;

	/*BR2PSI = BL1PSI = TL1PSI = SMALL;
	BR3PSI = BL2PSI = TL2PSI = LARGE;*/

	///* Search for Psi, Omega Over the Bottom Right corner of the domain "BR2, and BR3" */

	//for (int I = (NIM * 0.9); I < NIM; ++I)
	//{
	//	for (int J = 0; J < (NJM * 0.15); ++J)
	//	{
	//		int IJ = LI[I] + J;

	//		if (PP[IJ] < BR3PSI)
	//		{
	//			BR3PSI = PP[IJ];
	//			BR3I = I;
	//			BR3J = J;
	//			Unorth = U[IJ + 1];
	//			Usouth = U[IJ - 1];
	//			Veast = V[LI[I + 1] + J];
	//			Vwest = V[LI[I - 1] + J];
	//			DUDY = (Unorth - Usouth) * 0.5 * M;
	//			DVDX = (Veast - Vwest) * 0.5 * N;
	//			BR3OMEGA = DVDX - DUDY;
	//		}

	//		if (PP[IJ] > BR2PSI)
	//		{
	//			BR2PSI = PP[IJ];
	//			BR2I = I;
	//			BR2J = J;
	//			Unorth = U[IJ + 1];
	//			Usouth = U[IJ - 1];
	//			Veast = V[LI[I + 1] + J];
	//			Vwest = V[LI[I - 1] + J];
	//			DUDY = (Unorth - Usouth)*0.5*M;
	//			DVDX = (Veast - Vwest)*0.5*N;
	//			BR2OMEGA = DVDX - DUDY;
	//		}
	//	}
	//}

	//outputfile << "Strenght of The BR2 Vortex : "
	//	<< setw(12) << left << scientific << setprecision(6) << BR2PSI << endl;
	//outputfile << "Location X: " << setw(8) << left << fixed << setprecision(6) << BR2I*DY << endl;
	//outputfile << "Location Y: " << setw(8) << left << fixed << setprecision(6) << BR2J*DX << endl;
	//outputfile << "BR2OMEGA: " << setw(12) << left << scientific << setprecision(6) << BR2OMEGA << endl;
	//outputfile << endl << endl;

	//outputfile << "Strenght of The BR3 Vortex : "
	//	<< setw(12) << left << scientific << setprecision(6) << BR3PSI << endl;
	//outputfile << "Location X: " << setw(8) << left << fixed << setprecision(6) << BR3I*DY << endl;
	//outputfile << "Location Y: " << setw(8) << left << fixed << setprecision(6) << BR3J*DX << endl;
	//outputfile << "BR3OMEGA: " << setw(12) << left << scientific << setprecision(6) << BR3OMEGA << endl;
	//outputfile << endl << endl;

	//BL1PSI = TL1PSI = SMALL;
	//BL2PSI = TL2PSI = LARGE;

	///* Search for Psi, Omega Over the Bottom lef corner of the domain "BL1, and BL2" */

	//for (int I = 0; I < (NIM * 0.15); ++I)
	//{
	//	for (int J = 0; J < (NJM * 0.15); ++J)
	//	{
	//		int IJ = LI[I] + J;

	//		if (PP[IJ] < BL2PSI)
	//		{
	//			BL2PSI = PP[IJ];
	//			BL2I = I;
	//			BL2J = J;
	//			Unorth = U[IJ + 1];
	//			Usouth = U[IJ - 1];
	//			Veast = V[LI[I + 1] + J];
	//			Vwest = V[LI[I - 1] + J];
	//			DUDY = (Unorth - Usouth) * 0.5 * M;
	//			DVDX = (Veast - Vwest) * 0.5 * N;
	//			BL2OMEGA = DVDX - DUDY;
	//		}

	//		if (PP[IJ] > BL1PSI)
	//		{
	//			BL1PSI = PP[IJ];
	//			BL1I = I;
	//			BL1J = J;
	//			Unorth = U[IJ + 1];
	//			Usouth = U[IJ - 1];
	//			Veast = V[LI[I + 1] + J];
	//			Vwest = V[LI[I - 1] + J];
	//			DUDY = (Unorth - Usouth)*0.5*M;
	//			DVDX = (Veast - Vwest)*0.5*N;
	//			BL1OMEGA = DVDX - DUDY;
	//		}
	//	}
	//}

	//outputfile << "Strenght of The BL1 Vortex : "
	//	<< setw(12) << left << scientific << setprecision(6) << BL1PSI << endl;
	//outputfile << "Location X: " << setw(8) << left << fixed << setprecision(6) << BL1I*DY << endl;
	//outputfile << "Location Y: " << setw(8) << left << fixed << setprecision(6) << BL1J*DX << endl;
	//outputfile << "BL1OMEGA: " << setw(10) << left << fixed << setprecision(6) << BL1OMEGA << endl;
	//outputfile << endl << endl;

	//outputfile << "Strenght of The BL2 Vortex : "
	//	<< setw(12) << left << scientific << setprecision(6) << BL2PSI << endl;
	//outputfile << "Location X: " << setw(8) << left << fixed << setprecision(6) << BL2I*DY << endl;
	//outputfile << "Location Y: " << setw(8) << left << fixed << setprecision(6) << BL2J*DX << endl;
	//outputfile << "BL2OMEGA: " << setw(12) << left << scientific << setprecision(6) << BL2OMEGA << endl;
	//outputfile << endl << endl;

	//TL1PSI = SMALL;
	//TL2PSI = LARGE;

	///* Search for Psi, Omega Over the Bottom lef corner of the domain "TL1, and TL2" */

	//for (int I = 0; I < (NIM * 0.15); ++I)
	//{
	//	for (int J = (NJM * 0.9); J < NJM; ++J)
	//	{
	//		int IJ = LI[I] + J;

	//		if (PP[IJ] < TL2PSI)
	//		{
	//			TL2PSI = PP[IJ];
	//			TL2I = I;
	//			TL2J = J;
	//			Unorth = U[IJ + 1];
	//			Usouth = U[IJ - 1];
	//			Veast = V[LI[I + 1] + J];
	//			Vwest = V[LI[I - 1] + J];
	//			DUDY = (Unorth - Usouth) * 0.5 * M;
	//			DVDX = (Veast - Vwest) * 0.5 * N;
	//			TL2OMEGA = DVDX - DUDY;
	//		}

	//		if (PP[IJ] > TL1PSI)
	//		{
	//			TL1PSI = PP[IJ];
	//			TL1I = I;
	//			TL1J = J;
	//			Unorth = U[IJ + 1];
	//			Usouth = U[IJ - 1];
	//			Veast = V[LI[I + 1] + J];
	//			Vwest = V[LI[I - 1] + J];
	//			DUDY = (Unorth - Usouth)*0.5*M;
	//			DVDX = (Veast - Vwest)*0.5*N;
	//			TL1OMEGA = DVDX - DUDY;
	//		}
	//	}
	//}

	//outputfile << "Strenght of The TL1 Vortex : "
	//	<< setw(12) << left << scientific << setprecision(6) << TL1PSI << endl;
	//outputfile << "Location X: " << setw(8) << left << fixed << setprecision(6) << TL1I*DY << endl;
	//outputfile << "Location Y: " << setw(8) << left << fixed << setprecision(6) << TL1J*DX << endl;
	//outputfile << "TL1OMEGA: " << setw(10) << left << fixed << setprecision(6) << TL1OMEGA << endl;
	//outputfile << endl << endl;

	//outputfile << "Strenght of The TL2 Vortex : "
	//	<< setw(12) << left << scientific << setprecision(6) << TL2PSI << endl;
	//outputfile << "Location X: " << setw(8) << left << fixed << setprecision(6) << TL2I*DY << endl;
	//outputfile << "Location Y: " << setw(8) << left << fixed << setprecision(6) << TL2J*DX << endl;
	//outputfile << "TL2OMEGA: " << setw(12) << left << scientific << setprecision(6) << TL2OMEGA << endl;
	//outputfile << endl << endl;


	/*double *HORVORTICITY, *VERVORTICITY;
	HORVORTICITY = (double*)malloc(NIM*sizeof(double));
	VERVORTICITY = (double*)malloc(NJM*sizeof(double));*/


	///* Vorticity Valuse across a Horizontal line passing through the geometric center of the domain */

	//for (int I = 0; I < NIM; ++I)
	//{
	//	int IJ = LI(I) + (NJM / 2);

	//	Unorth = U(IJ + 1);
	//	Usouth = U(IJ - 1);
	//	Veast = V(LI(I + 1) + (NJM / 2));
	//	Vwest = V(LI(I - 1) + (NJM / 2));
	//	DUDY = (Unorth - Usouth) * 0.5 * M;
	//	DVDX = (Veast - Vwest) * 0.5 * N;
	//	HORVORTICITY(I) = DUDY - DVDX;
	//}

	///* Vorticity Valuse across a Vertical line passing through the geometric center of the domain */

	//for (int J = 0; J < NJM; ++J)
	//{
	//	int IJ = LI((NIM / 2)) + J;

	//	Unorth = U(IJ + 1);
	//	Usouth = U(IJ - 1);
	//	Veast = V(LI((NIM / 2) + 1)) + J;
	//	Vwest = V(LI((NIM / 2) - 1)) + J;
	//	DUDY = (Unorth - Usouth) * 0.5 * M;
	//	DVDX = (Veast - Vwest) * 0.5 * N;
	//	VERVORTICITY(J) = DVDX - DUDY;
	//}


	///* Vorticity Valuse across a Horizontal line passing through the geometric center of the domain */

	//outputfile << "Vorticity Valuse across a Horizontal line passing through the geometric center of the domain" << endl;

	//for (int I = 0; I < NIM; ++I)
	//{
	//	int IJ = LI(I) + (NJM / 2);

	//	outputfile << setw(5) << fixed << setprecision(3) << right << setfill('0') << I << "     "
	//		<< setw(18) << fixed << setprecision(12) << left << setfill('0') << HORVORTICITY(I) << "     " << endl << endl;

	//}

	///* Vorticity Valuse across a Vertical line passing through the geometric center of the domain */

	//outputfile << "Vorticity Valuse across a Vertical line passing through the geometric center of the domain" << endl << endl;

	//for (int J = 0; J < NJM; ++J)
	//{
	//	int IJ = LI((NIM / 2)) + J;

	//	outputfile << setw(5) << fixed << setprecision(3) << right << setfill('0') << J << "     "
	//		<< setw(18) << fixed << setprecision(12) << left << setfill('0') << VERVORTICITY(J) << "     " << endl << endl;
	//}



	free(X);
	free(XC);
	free(Y);
	free(YC);
	free(LI);
	free(U);
	free(V);
	free(P);
	free(T);
	free(PP);
	free(UO);
	free(VO);
	free(PO);
	free(TO);
	free(UOO);
	free(VOO);
	free(POO);
	free(TOO);
	free(AE);
	free(AW);
	free(AN);
	free(AS);
	free(AP);
	free(LW);
	free(LS);
	free(UN);
	free(UE);
	free(RES);
	free(LES);
	free(LPR);


	hipFree(D_RES);
	hipFree(D_LS);
	hipFree(D_LW);
	hipFree(D_UN);
	hipFree(D_UE);
	hipFree(D_LPR);

	hipDeviceReset();

	return 0;
}

